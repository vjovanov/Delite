
#include <hip/hip_runtime.h>
#include <sys/time.h>

struct timeval myprofiler_start, myprofiler_end;

void printTime(void) {
	printf("Time : %ld\n", ((myprofiler_end.tv_sec * 1000000 + myprofiler_end.tv_usec) - (myprofiler_start.tv_sec * 1000000 + myprofiler_start.tv_usec)));
}

void mytic(void) {
	hipDeviceSynchronize();
	gettimeofday(&myprofiler_start,NULL);
}

void mytoc(void) {
	hipDeviceSynchronize();
gettimeofday(&myprofiler_end,NULL);
printTime();
}

